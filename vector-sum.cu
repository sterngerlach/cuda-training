#include "hip/hip_runtime.h"

/* vector-sum.cu */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_CALL(call) \
    { \
        const hipError_t error = call; \
        \
        if (error != hipSuccess) { \
            fprintf(stderr, "Error (%s:%d), code: %d, reason: %s\n", \
                    __FILE__, __LINE__, \
                    error, hipGetErrorString(error)); \
                exit(EXIT_FAILURE); \
        } \
    }

void checkResult(float* hostResult, float* devResult, int vecSize)
{
    const double epsilon = 1.0e-8;

    int i;
    
    for (i = 0; i < vecSize; ++i) {
        if (fabsf(hostResult[i] - devResult[i]) > epsilon) {
            fprintf(stderr, "Results do not match!\n");
            fprintf(stderr, "Index %d, Host result: %5.2f, GPU result: %5.2f",
                    i, hostResult[i], devResult[i]);
            return;
        }
    }

    printf("Results match.\n");

    return;
}

void initializeVector(float* vec, int vecSize)
{
    int i;

    srand((unsigned int)time(NULL));

    for (i = 0; i < vecSize; ++i)
        vec[i] = (float)(rand() & 0xFF) / 10.0f;

    return;
}

void sumVectorsOnHost(float* vecA, float* vecB, float* vecC, int vecSize)
{
    int i;

    for (i = 0; i < vecSize; ++i)
        vecC[i] = vecA[i] + vecB[i];

    return;
}

__global__ void sumVectorsOnGPU(float* vecA, float* vecB, float* vecC)
{
    int i = threadIdx.x;

    vecC[i] = vecA[i] + vecB[i];
}

int main(int argc, char** argv)
{
    int dev;
    int numOfElements;
    size_t numOfBytes;

    float* hostVecA;
    float* hostVecB;
    float* hostVecC;
    float* devVecA;
    float* devVecB;
    float* devVecC;
    float* devResult;
    
    /* Setup device */
    dev = 0;
    CHECK_CUDA_CALL(hipSetDevice(dev));
    
    /* Set vector size */
    numOfElements = 32;
    numOfBytes = numOfElements * sizeof(float);
    printf("Vector size: %d\n", numOfElements);
    
    /* Allocate host memory */
    hostVecA = (float*)calloc(numOfElements, sizeof(float));
    hostVecB = (float*)calloc(numOfElements, sizeof(float));
    hostVecC = (float*)calloc(numOfElements, sizeof(float));
    devResult = (float*)calloc(numOfElements, sizeof(float));
    
    /* Initialize vectors */
    initializeVector(hostVecA, numOfElements);
    initializeVector(hostVecB, numOfElements);
    
    /* Allocate device memory */
    CHECK_CUDA_CALL(hipMalloc((float**)&devVecA, numOfBytes));
    CHECK_CUDA_CALL(hipMalloc((float**)&devVecB, numOfBytes));
    CHECK_CUDA_CALL(hipMalloc((float**)&devVecC, numOfBytes));

    /* Transfer vector data from host to device */
    CHECK_CUDA_CALL(hipMemcpy(devVecA, hostVecA, numOfBytes, hipMemcpyHostToDevice));
    CHECK_CUDA_CALL(hipMemcpy(devVecB, hostVecB, numOfBytes, hipMemcpyHostToDevice));
    CHECK_CUDA_CALL(hipMemcpy(devVecC, devResult, numOfBytes, hipMemcpyHostToDevice));

    /* Call kernel from host */
    dim3 block(numOfElements);
    dim3 grid(1);

    sumVectorsOnGPU<<<grid, block>>>(devVecA, devVecB, devVecC);
    CHECK_CUDA_CALL(hipDeviceSynchronize());

    printf("Execution configuration: <<<%d, %d>>>\n", grid.x, block.x);

    /* Check kernel error */
    CHECK_CUDA_CALL(hipGetLastError());

    /* Copy kernel result to host */
    CHECK_CUDA_CALL(hipMemcpy(devResult, devVecC, numOfBytes, hipMemcpyDeviceToHost));

    /* Add vectors in host to check device result */
    sumVectorsOnHost(hostVecA, hostVecB, hostVecC, numOfElements);

    /* Check device result */
    checkResult(hostVecC, devResult, numOfElements);

    /* Free device global memory */
    CHECK_CUDA_CALL(hipFree(devVecA));
    CHECK_CUDA_CALL(hipFree(devVecB));
    CHECK_CUDA_CALL(hipFree(devVecC));

    /* Free host memory */
    free(hostVecA);
    free(hostVecB);
    free(hostVecC);
    free(devResult);
    
    /* Reset device */
    CHECK_CUDA_CALL(hipDeviceReset());

    return EXIT_SUCCESS;
}

